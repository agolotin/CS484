#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h> 
#define BLOCKSIZE 1024
#define MAXIT 1
#define TOTROWS		(BLOCKSIZE*8)
#define TOTCOLS		(BLOCKSIZE*8)
#define NOTSETLOC       -1010101 // for cells that are not fixed
#define SETLOC			-1000110
#define EPSILON		0.1

#define QMAX(x,y) (((x) > (y))? (x): (y))


int *lkeepgoing;
float *iplate;
float *oplate;
float *fixed;
float *tmp;
int ncols, nrows;

double When();
void *Compute();


int main(int argc, char *argv[])
{
	double t0, tottime;
	ncols = TOTCOLS;
	nrows = TOTROWS;

	
	hipMalloc((void **) &lkeepgoing, nrows * ncols * sizeof(int));
	hipMalloc((void **) &iplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &oplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &fixed,  nrows * ncols * sizeof(float));
	printf("Memory allocated\n");

	t0 = When();
	/* Now proceed with the Jacobi algorithm */
	Compute();

	tottime = When() - t0;
	printf("Total Time is: %lf sec.\n", tottime);

	return 0;
}

__global__ void InitArrays(float *ip, float *op, float *fp, int *kp, int ncols, int nrows)
{
	int i;
	float *fppos, *oppos, *ippos;
	int *kppos;
	int blockOffset;
	int rowStartPos;
	int colsPerThread;

	// Calculate the offset of the row
    blockOffset = blockIdx.x * ncols; //what block I correspond to
    // Calculate our offset into the row
	rowStartPos = threadIdx.x * (ncols/blockDim.x);
    // The number of cols per thread
    colsPerThread = ncols/blockDim.x;

	ippos = ip + blockOffset + rowStartPos;
	fppos = fp + blockOffset + rowStartPos;
	oppos = op + blockOffset + rowStartPos;
	kppos = kp + blockOffset + rowStartPos;

	for (i = 0; i < colsPerThread; i++) {
		fppos[i] = NOTSETLOC; // Not Fixed
		ippos[i] = 50;
		oppos[i] = 50;
        kppos[i] = 1; // Keep Going
	}

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Insert code to set the rest of the boundary and fixed positions

	//Left and right columns have to be set to 0 
	fp[idx * ncols] = SETLOC;
	ip[idx * ncols] = 0;
	op[idx * ncols] = 0;
	kp[idx * ncols] = 0;

	fp[idx * ncols + (ncols - 1)] = SETLOC;
	ip[idx * ncols + (ncols - 1)] = 0;
	op[idx * ncols + (ncols - 1)] = 0;
	kp[idx * ncols + (ncols - 1)] = 0;

	//top row has to be set to 0
	if (idx == 0) {
		for (i = 0; i < colsPerThread; i++) {
			fppos[i] = SETLOC; // Fixed
			ippos[i] = 0;
			oppos[i] = 0;
			kppos[i] = 0; // Not Keep Going
		}
	}

	//bottom row has to be set to 100
	if (idx == nrows - 1) {
		for (i = 0; i < colsPerThread; i++) {
			fppos[i] = SETLOC; // Fixed
			ippos[i] = 100;
			oppos[i] = 100;
			kppos[i] = 0; // Not Keep Going
		}
	}
}
__global__ void doCalc(float *iplate, float *oplate, int ncols)
{
	/* Compute the 5 point stencil for my region */
	extern __shared__ float matrix[];
	
	int i, j, row, top_row, bottom_row;
	int begin, end;

	int iproc = blockIdx.x;
	int nproc = gridDim.x;

	int nrows = TOTROWS;

	begin = iproc * nrows/nproc;
	end = begin + nrows/nproc;

	float* shared_rowup = &matrix[0];
	float* shared_rowcur = &matrix[TOTCOLS];
	float* shared_rowdown = &matrix[2*TOTCOLS];

	// Adjust boundary values
	if (iproc == 0) {
		begin = begin + 1;
	}
	if (iproc == (nproc - 1)) {
		end = nrows - 1;
	}

	// Load top and current prev row to global memory
	// Bottom row can only be accessed later
	for (j = threadIdx.x; j < ncols; j += blockDim.x) {
		shared_rowup[j] = oplate[(begin-1)*ncols + j];
		shared_rowcur[j] = oplate[begin * ncols + j];
	}


	for (i = begin; i < end; i++) {
		row = i * ncols;
		top_row = (i-1) * ncols;
		bottom_row = (i+1) * ncols;
		for (j = threadIdx.x; j < ncols; j += blockDim.x) {
			shared_rowdown[j] = oplate[bottom_row + j];
		}
		__syncthreads();

		//perform calculation from shared memory
		for (j = threadIdx.x; j < ncols; j += blockDim.x) {
			if (j > 0 && j < ncols - 1) {
				iplate[row + j] = (4 * shared_rowcur[j] + shared_rowup[j] + shared_rowdown[j] 
								+ shared_rowcur[j-1] + shared_rowcur[j+1]) * 0.125;
			}
		}
	}
}

__global__ void doCheck(float *iplate, float *oplate, float *fixed, int *lkeepgoing, int ncols)
{
	// Calculate keepgoing array
	int i, j, row, top_row, bottom_row;
	float *cur_pos_val, *up_val, *down_val;
	int begin, end;
	int nrows;

	float maxerror, error;

	extern __shared__ float* maxerrorlist;

	int iproc = blockIdx.x;
	int nproc = gridDim.x;

	ncols = TOTCOLS;
	nrows = TOTROWS;

	begin = iproc * nrows/nproc;
	end = begin + nrows/nproc;

	// Adjust boundary values
	if (iproc == 0)
		begin = begin + 1;
	if (iproc == (nproc - 1))
		end = nrows - 1;

	for (i = begin; i < end; i++)
	{
		row = i * ncols;
		top_row = (i-1) * ncols;
		bottom_row = (i+1) * ncols;


		cur_pos_val = &(iplate[row + threadIdx.x]);
		up_val =  &(iplate[top_row+ threadIdx.x]);
		down_val =  &(iplate[bottom_row + threadIdx.x]);

		for (j = threadIdx.x; j < ncols && maxerror <= EPSILON; j+=blockDim.x)
		{
			if (fixed[row + j] == NOTSETLOC)
			{
				error = *currpos - (*upptr + *dnptr + *(currpos -1) + *(currpos +1)) * 0.25;
				
				if (maxerrorlist[threadIdx.x] < error) {
					maxerrorist[threadIdx.x] = error;
				{
			}


		}
	}
	// do reduction in shared mem
	int q;
	for(q = blockDim.x/2; q > 0; q >>= 1) //I don't know if it's gonna work...
	{
		if (threadIdx.x < q)
		{
			if (maxerrorlist[threadIdx.x] > maxerrorlist[threadIdx.x+q]) {
				maxerrorlist[threadIdx.x] = maxerrorlist[threadIdx.x+q];
			}
		}
		__syncthreads();
	}
	maxerror = maxerrorlist[0];

	if (threadIdx.x == 0)
	{
		/* If the maxerror > MAXERROR allowed, I must keep going */
		if (maxerror > EPSILON)
			lkeepgoing[iproc] = 1;
		else 
			lkeepgoing[iproc] = 0;
	
		//fprintf(stderr,"%d: maxerror %f -- stopped checking at [%d, %d]\n", iproc, maxerror, i, j);
	}
}

//__global__ void reduceSingle(int *idata, int *single, int nrows) //bad 
//{
//	int i;
//	if(threadIdx.x == 0) {
//		*single = 0;
//		for(i = 0; i < nrows; i++) {
//			*single += idata[i];
//		}
//		printf(" end %d\n",*single);
//	}
//}

//__global__ void reduceSingleSequentialAddressing(int *idata, int *single, int nrows)
//{
//	// Reduce rows to the first element in each row
//	int i;
//    int rowStartPos;
//    int colsPerThread;
//	extern __shared__ int parts[]; // shared array that holds temporary sums
//	
//    // A block gets a row, each thread will reduce part of a row
//
//    // Calculate our offset into the row
//	rowStartPos = threadIdx.x * (nrows/blockDim.x); //number of rows / number of threads in a block
//    // The number of cols per thread
//    colsPerThread = nrows/blockDim.x; //number of rows / block dimention = 8k/1024 = 8
//
//	int tid = threadIdx.x;
//	// Sum my part of 1D array and put it in shared memory
//	parts[tid] = 0;
//	for (i = rowStartPos; i < colsPerThread+rowStartPos; i++) {
//		parts[tid] += idata[i];
//	}
//
//	//sequential addressing method
//	if (tid < TOTROWS / 2) { parts[tid] += parts[tid + TOTROWS / 2]; }
//	__syncthreads(); 
//
//	if (tid < TOTROWS / 4) { parts[tid] += parts[tid + TOTROWS / 4]; }
//	__syncthreads(); 
//
//	if (tid < TOTROWS / 8) { parts[tid] += parts[tid + TOTROWS / 8]; }
//	__syncthreads(); 
//
//	if (tid < TOTROWS / 16) { parts[tid] += parts[tid + TOTROWS / 16]; }
//	__syncthreads(); 
//
//	if (tid < TOTROWS / 32) { parts[tid] += parts[tid + TOTROWS / 32]; }
//	__syncthreads(); 
//
//	if(threadIdx.x == 0) {
//		*single = 0;
//		for(i = 0; i < 32; i++) {
//			*single += parts[i];
//		}
//	}
//}


__global__ void reduceSingle(int *idata, int *single, int nrows) {
	// Reduce rows to the first element in each row
	int i;
    int rowStartPos;
    int colsPerThread;
	extern __shared__ int parts[]; // shared array that holds temporary sums
	
    // A block gets a row, each thread will reduce part of a row

    // Calculate our offset into the row
	rowStartPos = threadIdx.x * (nrows/blockDim.x); //number of rows / number of threads in a block
    // The number of cols per thread
    colsPerThread = nrows/blockDim.x; //number of rows / block dimention = 8k/1024 = 8

	// Sum my part of 1D array and put it in shared memory
	parts[threadIdx.x] = 0;
	for (i = threadIdx.x; i < nrows; i+=blockDim.x) { // everyone will start in a chunk together grabbing a chunk at a time and processing it later
		parts[threadIdx.x] += idata[i];
	}

	int tid = threadIdx.x;
	if (tid < TOTROWS / 2) { parts[tid] += parts[tid + TOTROWS / 2]; }
	__syncthreads(); 

	if (tid < TOTROWS / 4) { parts[tid] += parts[tid + TOTROWS / 4]; }
	__syncthreads(); 

	if (tid < TOTROWS / 8) { parts[tid] += parts[tid + TOTROWS / 8]; }
	__syncthreads(); 

	if (tid < TOTROWS / 16) { parts[tid] += parts[tid + TOTROWS / 16]; }
	__syncthreads(); 

	if (tid < TOTROWS / 32) { parts[tid] += parts[tid + TOTROWS / 32]; }
	__syncthreads(); 

	if(threadIdx.x == 0) {
		*single = 0;
		for(i = 0; i < 32; i++) {
			*single += parts[i];
		}
	}
}

__global__ void reduceSum(int *idata, int *odata, unsigned int ncols)
{
	// Reduce rows to the first element in each row
	int i;
    int blockOffset;
    int rowStartPos;
    int colsPerThread;
    int *mypart;
	
    // Each block gets a row, each thread will reduce part of a row

	// Calculate the offset of the row
    blockOffset = blockIdx.x * ncols;
    // Calculate our offset into the row
	rowStartPos = threadIdx.x * (ncols/blockDim.x);
    // The number of cols per thread
    colsPerThread = ncols/blockDim.x;

	mypart = idata + blockOffset + rowStartPos;

	// Sum all of the elements in my thread block and put them 
    // into the first column spot
	for (i = 1; i < colsPerThread; i++) {
		mypart[0] += mypart[i];
	}
	__syncthreads(); // Wait for everyone to complete
        // Now reduce all of the threads in my block into the first spot for my row
	if(threadIdx.x == 0) {
		odata[blockIdx.x] = 0;
		for(i = 0; i < blockDim.x; i++) {
			odata[blockIdx.x] += mypart[i*colsPerThread];
		}
	}
	// We cant synchronize between blocks, so we will have to start another kernel
}
	
void *Compute()
{
	printf("Entered compute\n");
	int *keepgoing_single;
	int *keepgoing_sums;
	int keepgoing;
	int blocksize = BLOCKSIZE;
	int iteration;

	ncols = TOTCOLS;
	nrows = TOTROWS;

	printf("About to init arrays\n");
	// One block per row
	InitArrays<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols, nrows);


	float** plate = (float**)malloc(nrows * ncols * sizeof(float));
	hipMemcpy(plate, iplate, sizeof(float)*nrows*ncols, hipMemcpyDeviceToHost);
	int q,w;
	for (q = 0; q < nrows; q++) {
		for (w = 0; w < ncols; w++) {
			printf("%f, ", plate[q][w]);
		}
		printf("\n");
	}
	exit(-1);


	hipMalloc((void **)&keepgoing_single, 1 * sizeof(int));
	keepgoing = 1;
	hipMalloc((void **)&keepgoing_sums, nrows * sizeof(int));
 	int *peek = (int *)malloc(nrows*sizeof(int));

	for (iteration = 0; (iteration < MAXIT) && keepgoing; iteration++)
	{
		doCalc<<< nrows, blocksize, TOTROWS*TOTCOLS*sizeof(double)>>>(iplate, oplate, ncols);
		doCheck<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols);
		reduceSum<<< nrows, blocksize>>>(lkeepgoing, keepgoing_sums, ncols);
		hipMemcpy(peek, keepgoing_sums, nrows*sizeof(int), hipMemcpyDeviceToHost);
		printf("after hipMemcpy \n");
		int i;
 		for(i = 0; i < nrows; i++) {
			printf("%d, ",peek[i]);
		}
		// Now we have the sum for each row in the first column, 
		//  reduce to one value
		double t0 = When();
		int timeit;
	//	for (timeit=0; timeit< 10000; timeit++) {
	//		//reduceSingle<<<1, blocksize>>>(keepgoing_sums, keepgoing_single, nrows); // for the bad one 
	//		reduceSingle<<<1, blocksize, blocksize*sizeof(int)>>>(keepgoing_sums, keepgoing_single, nrows); //third argument is for how much memory to give to dynamic __shared__ array
	//	}
		double endit = When() - t0;
		keepgoing = 0;
		hipMemcpy(&keepgoing, keepgoing_single, 1 * sizeof(int), hipMemcpyDeviceToHost);
		printf("keepgoing = %d, time 10000 = %f\n", keepgoing, endit);

		/* swap the new value pointer with the old value pointer */
		tmp = oplate;
		oplate = iplate;
		iplate = tmp;
	}
	free(peek);
	hipFree(keepgoing_single);
	hipFree(keepgoing_sums);
	printf("Finished in %d iterations\n", iteration);
}

/* Return the current time in seconds, using a double precision number.       */
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}
