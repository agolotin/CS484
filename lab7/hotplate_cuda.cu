#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_ROWS 8192
#define NUM_COLS 8192
#define EPSILON  0.1
#define TRUE	1.0f
#define FALSE	0.0f
#define TOTITERATIONS 359

double When()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}


__global__ void initArrays(float *from, float *to, float *locked, int size) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (idx < size) {
		// inner is 50s
		if ((blockIdx.x > 0) && (blockIdx.x < NUM_ROWS)){
			if ((threadIdx.x > 0) && (threadIdx.x < NUM_COLS)) {
				from[idx] = 50;
				to[idx] = 50;
				locked[idx] = FALSE;
			}
		}
		
		// sides are 0
		if ((threadIdx.x == 0) || (threadIdx.x == NUM_COLS-1)) {
			from[idx] = 0;
			to[idx] = 0;
			locked[idx] = TRUE;
		}
		
		// top is 0
		if (blockIdx.x == 0) {
			from[idx] = 0;
			to[idx] = 0;
			locked[idx] = TRUE;
		}
		
		// bottom is 100
		if (blockIdx.x == NUM_ROWS-1) {
			from[idx] = 100;
			to[idx] = 100;
			locked[idx] = TRUE;
		}
	}
}

__global__ void resetKeepgoing(int *lkeepgoing, int *keepgoing) {
	lkeepgoing[threadIdx.x] = 1;
	if (threadIdx.x == 0)
		*keepgoing = 0;
}


__global__ void calculate(float *from, float *to, float* locked, int size, int *lkeepgoing) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	float total, self;
	
	if (idx < size) {
		if (locked[idx] == TRUE) {
			return;
		}
		total = from[idx - NUM_COLS] + from[idx-1] + from[idx + 1] + from[idx + NUM_COLS];
		self = from[idx];
		
		to[idx] = (total + 4 * self) * 0.125;
		
		// Set the keepgoing data for the block
		if ((fabs(self - (total)/4) < EPSILON)) {
			lkeepgoing[blockIdx.x] = 0;
		}
		
	}
}

__global__ void reduceSingle(int *lkeepgoing, int *keepgoing)
{
    extern __shared__ int sdata[];
	unsigned int tid, i, s;

    // Calculate our offset into the row
	int rowStartPos = threadIdx.x * (NUM_ROWS/blockDim.x); //number of rows / number of threads in a block
    // The number of cols per thread
    int colsPerThread = NUM_ROWS/blockDim.x; //number of rows / block dimention = 8k/1024 = 8

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    tid = threadIdx.x;

	// Sum my part of 1D array and put it in shared memory
	// Method 1
	sdata[tid] = 0;
	for (i = rowStartPos; i < colsPerThread+rowStartPos; i++) {
		sdata[tid] += lkeepgoing[i];
	}
	__syncthreads();
//    i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
//    sdata[tid] = lkeepgoing[i] & lkeepgoing[i+blockDim.x];
//    __syncthreads();

	if (tid < NUM_ROWS / 2) 
	{
        sdata[tid] &= sdata[tid + 4096]; __syncthreads();
        sdata[tid] &= sdata[tid + 2048]; __syncthreads();
        sdata[tid] &= sdata[tid + 1024]; __syncthreads();
        sdata[tid] &= sdata[tid +  512]; __syncthreads();
        sdata[tid] &= sdata[tid +  256]; __syncthreads();
        sdata[tid] &= sdata[tid +  128]; __syncthreads();
        sdata[tid] &= sdata[tid +   64]; __syncthreads();
        sdata[tid] &= sdata[tid +   32]; __syncthreads();
        sdata[tid] &= sdata[tid +   16]; __syncthreads();
        sdata[tid] &= sdata[tid +    8]; __syncthreads();
        sdata[tid] &= sdata[tid +    4]; __syncthreads();
        sdata[tid] &= sdata[tid +    2]; __syncthreads();
        sdata[tid] &= sdata[tid +    1]; __syncthreads();
	}

	// Method 2
//	sdata[tid] = 0;
//	for (i = tid; i < NUM_ROWS; i+=blockDim.x) { // everyone will start in a chunk together grabbing a chunk at a time and processing it later
//		sdata[threadIdx.x] += lkeepgoing[i];
//	}
//	__syncthreads();
//
//	if (tid < NUM_ROWS / 2) 
//	{
//        sdata[tid] &= sdata[tid + 4096]; __syncthreads();
//        sdata[tid] &= sdata[tid + 2048]; __syncthreads();
//        sdata[tid] &= sdata[tid + 1024]; __syncthreads();
//        sdata[tid] &= sdata[tid +  512]; __syncthreads();
//        sdata[tid] &= sdata[tid +  256]; __syncthreads();
//        sdata[tid] &= sdata[tid +  128]; __syncthreads();
//        sdata[tid] &= sdata[tid +   64]; __syncthreads();
//        sdata[tid] &= sdata[tid +   32]; __syncthreads();
//        sdata[tid] &= sdata[tid +   16]; __syncthreads();
//        sdata[tid] &= sdata[tid +    8]; __syncthreads();
//        sdata[tid] &= sdata[tid +    4]; __syncthreads();
//        sdata[tid] &= sdata[tid +    2]; __syncthreads();
//        sdata[tid] &= sdata[tid +    1]; __syncthreads();
//	}

      // Method 3
//    i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
//    sdata[tid] = lkeepgoing[i] & lkeepgoing[i+blockDim.x];
//    __syncthreads();
//	
//    // do reduction in shared memory
//    for(s=blockDim.x/2; s>32; s>>=1)
//    {
//        if (tid < s)
//        {
//            sdata[tid] &= sdata[tid + s];
//        }
//        __syncthreads();
//    }
//	
//    if (tid < 32)
//    {
//        sdata[tid] &= sdata[tid + 32]; __syncthreads();
//        sdata[tid] &= sdata[tid + 16]; __syncthreads();
//        sdata[tid] &= sdata[tid +  8]; __syncthreads();
//        sdata[tid] &= sdata[tid +  4]; __syncthreads();
//        sdata[tid] &= sdata[tid +  2]; __syncthreads();
//        sdata[tid] &= sdata[tid +  1]; __syncthreads();
//    }
	
    // write result for this block to global mem
    if (tid == 0) *keepgoing = sdata[0];
}



int main(void) {
	double timestart, timefinish, timetaken; // host data
	float *from_d, *to_d, *locked;	// device data
	float *temp_d;
	int *lkeepgoing, *keepgoing;	// more device data
	int nBytes;
	int iterations;
	int SIZE, blocks, threadsperblock;
	int *steadyState;

	SIZE = NUM_ROWS * NUM_COLS;
	blocks = 8192;
	threadsperblock = 8192;

	steadyState = (int*)malloc(sizeof(int));
	*steadyState = 0;

	nBytes = SIZE*sizeof(float);
	hipMalloc((void **) &from_d, nBytes);
	hipMalloc((void **) &to_d, nBytes);
	hipMalloc((void **) &locked, nBytes);
	hipMalloc((void **) &lkeepgoing, blocks * sizeof(int));
	hipMalloc((void **) &keepgoing, sizeof(int));

	initArrays<<<blocks,threadsperblock>>> (from_d, to_d, locked, SIZE);
	
	iterations = 0;
	timestart = When();
	while (!*steadyState) {//&& TOTITERATIONS != iterations) {
		
		resetKeepgoing<<<1,blocks>>> (lkeepgoing, keepgoing);
		calculate<<<blocks,threadsperblock>>> (from_d, to_d, locked, SIZE, lkeepgoing);
		reduceSingle<<<1,blocks, blocks*sizeof(int)>>> (lkeepgoing, keepgoing);
	
		hipMemcpy(steadyState, keepgoing, sizeof(int), hipMemcpyDeviceToHost);

		iterations++;
		temp_d = from_d;
		from_d = to_d;
		to_d = temp_d;
		
		printf("Iteration %d\n", iterations);
	}

	timefinish = When();
	float* plate = (float*)malloc(sizeof(float) * SIZE);
	hipMemcpy(plate, to_d, sizeof(float)*SIZE, hipMemcpyDeviceToHost);

	hipFree(from_d); 
	hipFree(lkeepgoing);
	hipFree(keepgoing);
	free(steadyState);

	timetaken = timefinish - timestart;

	printf("Iteration %d time %f\n", iterations, timetaken);
	int k;
	for (k = 0; k < SIZE; k++) {
		if (k % 8191 == 0) {
			printf("\n");
		}
		printf("%d\t", plate[k]);
	}
	
	hipFree(to_d);
	free(plate);

	return 0;
}
