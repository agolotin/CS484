#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 5 //number of thread blocks
#define THREAD_NUM 5 //number of threads per block
#define STEP_SIZE 10000

__global__ void calculate(float* sum, int nbin, int step, int nthreads, int nblocks) {
	float x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	for (i = idx; i < nbin; i += nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

int main(void)
{
	float *sum_device, *sum_host;
	float step = 1.0/STEP_SIZE;

	size_t size = THREAD_NUM * BLOCK_SIZE * sizeof(float);
	sum_host = (float*)malloc(size);
	hipMalloc((void**) &sum_device, size);
	//set the sum_device to 0
	hipMemset(sum_device, 0, size);
	int block_size = BLOCK_SIZE;
	int grid_size = THREAD_NUM;
	calculate<<<block_size, grid_size>>>(sum_device, STEP_SIZE, step, THREAD_NUM, BLOCK_SIZE);
	//copy from device to host
	hipMemcpy(sum_host, sum_device, size, hipMemcpyDeviceToHost);
	int tid;
	float pi = 0;
	for (tid = 0; tid < THREAD_NUM * BLOCK_SIZE; tid++) 
		pi += sum_host[tid];
	pi *= step;

	printf("PI = %f\n", pi);
	free(sum_host);
	hipFree(sum_device);

	return EXIT_SUCCESS;
}
