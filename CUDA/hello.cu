
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}
